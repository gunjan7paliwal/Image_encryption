//#include "stdafx.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"



struct pixel
{  
    unsigned char r;
    unsigned char g;
    unsigned char b;
  
};
__device__
int power(int a,int b){
   for (int i=0;i<b;i++){
       a*=a;     
   }
   return a; 
}

__device__
void hamming_code(long *a)
{
    long z=*a;
    int d1,d2,d3,d4,d6,d7,d8,p1,p2,p3,tmp1,tmp2;
    tmp2=power(10,7);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d1=tmp2;
    tmp2=power(10,6);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d2=tmp2;
    tmp2=power(10,5);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d3=tmp2;
    tmp2=power(10,4);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d4=tmp2;
    tmp2=power(10,2);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d6=tmp2;
    tmp2=power(10,1);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d7=tmp2;
    tmp2=(*a)%10;
    d8=tmp2;

    p1=d1^d2^d3;
    p2=d1^d2^d4;
    p3=d1^d3^d4;
    *a=(*a)-100*d6-10*d7-d8+100*p1+10*p2+p3;
return ;
}

__device__
void bit_rotation(long *a)
{
    long z=*a,tmp,tmp1,tmp2,p1,p2,p3;
    tmp1=z/power(10,2);
    tmp2=tmp1%10;
    p1=tmp2;
    tmp1=z/power(10,1);
    tmp2=tmp1%10;
    p2=tmp2;
    tmp2=*a%10;
    p3=tmp2;
    *a=(*a)-100*p1-10*p2-p3;
    tmp=p1;
    p1=p2;
    p2=p3;
    p3=tmp;
    *a=(*a)+100*p1+10*p2+p3;
    return;
}

__device__
void rev_bit_rotation(long *a)
{
    long z=*a,tmp1,tmp2,p1,p2,p3,tmp;
    tmp1=z/power(10,2);
    tmp2=tmp1%10;
    p1=tmp2;
    tmp1=z/power(10,1);
    tmp2=tmp1%10;
    p2=tmp2;
    tmp2=*a%10;
    p3=tmp2;
    *a=(*a)-100*p1-10*p2-p3;
    tmp=p3;
    p3=p2;
    p2=p1;
    p1=tmp;
    //printf("%d\n",*a);
    *a=(*a)+100*p1+10*p2+p3;
    //printf("%d",*a);
    return;
}

__device__
void Torus_Auromorphism(int *a,int *b,int c)
{
    //k=1
    int x,y;
    x = (*a+*b)%c;
    y= (*a + 2*(*b))%c;
 
    *a=x;*b=y;
    return;
}

__device__
void Anti_Torus(int *a,int *b,int c)
{
    int x,y;
    x=(2*(*a)+(-1)*(*b)+100000*c)%c;
    y=((-1)*(*a)+*b+10000*c)%c;
    //printf("a=%d b=%d x=%d y=%d\n",*a,*b,x,y);
    *a=x;*b=y;
    //if(*a=2&&*b2)
    
    //*a=x;*b=y;
}



__device__
int DecToBinary(int num)
{
    int count=0, remainder, base = 1, binary = 0, no_of_1s = 0;
    
    while (num > 0)
    {
        count++;
        remainder = num % 2;
        if (remainder == 1)
        {
            no_of_1s++;
        }
        binary = binary + remainder * base;
        num = num / 2;
        base = base * 10;

    }
    binary+=100000000;

//    printf("binary = %d\n",binary);
    return binary;
}

__device__
int BinToDec(int num){
    num-=100000000;
    int dec=0,k=1,i=0;
    //printf("%d",num);

    while(1){
        
        dec=dec + k*(num%10);
        if(i==0)
            i=1;
        else
            i=2*i;
        if(num==0)
        {
            break;
        }

        k=2*i;
        
        num/=10;
    }
    //printf("\n%d",dec);
    return dec;
}

//Converting rgba image to greyscale
__global__
void rgba_to_greyscale(struct pixel* rgbaImage, struct pixel* greyImage,int w,int h)
{
     //int threadId = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
  // Calculate the row # of the d_Pin and d_Pout element
    int Col = blockIdx.x*blockDim.x + threadIdx.x;
    // Calculate the column # of the d_Pin and d_Pout element
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    // each thread computes one element of d_Pout if in range
    
    int threadId=Row*w+Col;


  if (threadId < w*h){
    const unsigned char R = rgbaImage[threadId].r;
    const unsigned char G = rgbaImage[threadId].g;
    const unsigned char B = rgbaImage[threadId].b;
    //greyimage[threadId]
    //__syncthreads();
    greyImage[threadId].r=  R/2;
    greyImage[threadId].g=  G/5;
    greyImage[threadId].b=B/1;
  }

}

__global__
void PictureKernel(struct pixel* IPimage, struct pixel* OPimage,long* R,long* G,long* B,int w,int h)
{
    // Calculate the row # of the d_Pin and d_Pout element
    int Col = blockIdx.x*blockDim.x + threadIdx.x;
    // Calculate the column # of the d_Pin and d_Pout element
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    // each thread computes one element of d_Pout if in range
    
    int tmp1,tmp2;
    tmp2=Col;tmp1=Row;
    Anti_Torus(&tmp1,&tmp2,w);
    
    //Anti Torus
    OPimage[tmp1*w+tmp2]=IPimage[Row*w+Col];
    


    R[Col+Row*w]=DecToBinary(IPimage[Col+Row*w].r);
    G[Col+Row*w]=DecToBinary(IPimage[Col+Row*w].g);
    B[Col+Row*w]=DecToBinary(IPimage[Col+Row*w].b);


    rev_bit_rotation(&R[Col+Row*w]);rev_bit_rotation(&G[Col+Row*w]);rev_bit_rotation(&B[Col+Row*w]);

    IPimage[Col+Row*w].r=BinToDec(R[Col+Row*w]);
    IPimage[Col+Row*w].g=BinToDec(G[Col+Row*w]);
    IPimage[Col+Row*w].b=BinToDec(B[Col+Row*w]);



    return;
}



int main(void)
{
    int i, w, h;
    char blah[3];

    FILE *f, *f2, *f3;
    
   

    f=fopen("pllencrypted.ppm", "rb");
    f2=fopen("plldecrypted.ppm", "wb"); 
    f3=fopen("decryptedrgb.ppm","wb");
    fscanf(f, "%s\n", blah);
    fscanf(f, "%d %d\n", &w, &h);
    fscanf(f, "%d\n", &i);
    struct pixel image[h][w],h_GreyImage[h][w]; 
    fread(&image, sizeof(image), 1, f);
    
  
   // clock_t end = clock();
    //double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    //printf("%f",time_spent);

    //long R[h][w],G[h][w],B[h][w];
    struct pixel *d_A,*d_F,*d_GreyImage;
    long *d_R,*d_G,*d_B;
    
    long n=w*h;
    //const long size=n;
    const long bytes = 3*sizeof(unsigned char)*n;

    //clock_t begin = clock();
 
    //Assigning memory in device

    hipMalloc((void **)&d_A,sizeof(pixel)*n);
    hipMalloc((void **)&d_F,sizeof(pixel)*n);
    hipMalloc((void **)&d_R,sizeof(long)*n);
    hipMalloc((void **)&d_G,sizeof(long)*n);
    hipMalloc((void **)&d_B,sizeof(long)*n);
     hipMalloc((void **)&d_GreyImage,sizeof(pixel)*n);

    hipMemcpy(d_A,image,bytes,hipMemcpyHostToDevice);
    clock_t begin = clock();

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(w/threadsPerBlock.x,h/threadsPerBlock.y); 

    PictureKernel<<<numBlocks,threadsPerBlock>>>(d_A,d_F,d_R,d_G,d_B,w,h);    

    
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("%f microseconds\n",time_spent*1000000);
    
    hipMemcpy(image,d_F,bytes,hipMemcpyDeviceToHost);
    hipMemcpy(d_A,image,bytes,hipMemcpyHostToDevice);
    rgba_to_greyscale<<<numBlocks,threadsPerBlock>>>(d_A,d_GreyImage,w,h);
    hipMemcpy(h_GreyImage,d_GreyImage,bytes,hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_F);
    hipFree(d_R);
    hipFree(d_G);
    hipFree(d_B);



    fprintf(f2, "%s\n", blah);
    fprintf(f2, "%d %d\n", w, h);
    fprintf(f2, "%d\n", 255);
    fprintf(f3, "%s\n", blah);
    fprintf(f3, "%d %d\n", w, h);
    fprintf(f3, "%d\n", 255);
    
    fwrite(&image, sizeof(image), 1, f2);
    fwrite(&h_GreyImage,sizeof(h_GreyImage),1,f3);
    fclose(f);
    fclose(f2);
    fclose(f3);
    return 0;
}

