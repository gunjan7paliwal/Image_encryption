//#include "stdafx.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"



struct pixel
{  
    unsigned char r;
    unsigned char g;
    unsigned char b;
  
};
__device__
int power(int a,int b){
    long sum=1;
   for (int i=1;i<=b;i++){
       sum=sum*a;     
   }
   return sum; 
}

__device__
void hamming_code(long *a)
{
    long z=*a;
    int d1,d2,d3,d4,d6,d7,d8,p1,p2,p3,tmp1,tmp2;
    tmp2=power(10,7);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d1=tmp2;
    tmp2=power(10,6);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d2=tmp2;
    tmp2=power(10,5);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d3=tmp2;
    tmp2=power(10,4);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d4=tmp2;
    tmp1=z/100;
    tmp2=tmp1%10;
    d6=tmp2;
    tmp1=z/10;
    tmp2=tmp1%10;
    d7=tmp2;
    tmp2=(*a)%10;
    d8=tmp2;

    p1=d1^d2^d3;
    p2=d1^d2^d4;
    p3=d1^d3^d4;
    *a=(*a)-100*d6-10*d7-d8+100*p1+10*p2+p3;
return ;
}
__device__
void bit_rotation(long *a)
{
    long z=*a,tmp,tmp1,tmp2,p1,p2,p3;
    tmp1=z/100;
    tmp2=tmp1%10;
    p1=tmp2;
    tmp1=z/10;
    tmp2=tmp1%10;
    p2=tmp2;
    tmp2=*a%10;
    p3=tmp2;
    *a=(*a)-100*p1-10*p2-p3;
    tmp=p1;
    p1=p2;
    p2=p3;
    p3=tmp;
    *a=(*a)+100*p1+10*p2+p3;
    return;
}

__device__
void rev_bit_rotation(long *a)
{
    long z=*a,tmp1,tmp2,p1,p2,p3,tmp;
    tmp1=z/power(10,2);
    tmp2=tmp1%10;
    p1=tmp2;
    tmp1=z/power(10,1);
    tmp2=tmp1%10;
    p2=tmp2;
    tmp2=*a%10;
    p3=tmp2;
    *a=(*a)-100*p1-10*p2-p3;
    tmp=p3;
    p3=p2;
    p2=p1;
    p1=tmp;
    //printf("%d\n",*a);
    *a=(*a)+100*p1+10*p2+p3;
    //printf("%d",*a);
    return;
}


__device__
void Torus_Auromorphism(int *a,int *b,int c)
{
    //k=1
    int x,y;
    x = (*a+*b)%c;
    y= (*a + 2*(*b))%c;
 
    *a=x;*b=y;
    return;
}

__device__
void Anti_Torus(int *a,int *b,int c)
{
    int x,y;
    x=(2*(*a)+(-1)*(*b)+100000*c)%c;
    y=((-1)*(*a)+*b+10000*c)%c;
    //printf("a=%d b=%d x=%d y=%d\n",*a,*b,x,y);
    *a=x;*b=y;
    //if(*a=2&&*b2)
    
    //*a=x;*b=y;
}


__device__
int DecToBinary(int num)
{
    int count=0, remainder, base = 1, binary = 0, no_of_1s = 0;
    
    while (num > 0)
    {
        count++;
        remainder = num % 2;
        if (remainder == 1)
        {
            no_of_1s++;
        }
        binary = binary + remainder * base;
        num = num / 2;
        base = base * 10;

    }
    binary+=100000000;

//    printf("binary = %d\n",binary);
    return binary;
}

__device__
int BinToDec(int num){
    num-=100000000;
    int dec=0,k=1,i=0;
    //printf("%d",num);

    while(1){
        
        dec=dec + k*(num%10);
        if(i==0)
            i=1;
        else
            i=2*i;
        if(num==0)
        {
            break;
        }

        k=2*i;
        
        num/=10;
    }
    //printf("\n%d",dec);
    return dec;
}



__global__
void PictureKernel(struct pixel* IPimage, struct pixel* OPimage,long* R,long* G,long* B,int w,int h)
{
    // Calculate the row # of the d_Pin and d_Pout element
    int Col = blockIdx.x*blockDim.x + threadIdx.x;
    // Calculate the column # of the d_Pin and d_Pout element
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    // each thread computes one element of d_Pout if in range
    
    int tmp1,tmp2;
    tmp2=Col;tmp1=Row;
    Torus_Auromorphism(&tmp1,&tmp2,w);

    R[Col+Row*w]=DecToBinary(IPimage[Col+Row*w].r);
    G[Col+Row*w]=DecToBinary(IPimage[Col+Row*w].g);
    B[Col+Row*w]=DecToBinary(IPimage[Col+Row*w].b);

    hamming_code(&R[Col+Row*w]);hamming_code(&G[Col+Row*w]);hamming_code(&B[Col+Row*w]);
    bit_rotation(&R[Col+Row*w]);bit_rotation(&G[Col+Row*w]);bit_rotation(&B[Col+Row*w]);


    IPimage[Col+Row*w].r=BinToDec(R[Col+Row*w]);
    IPimage[Col+Row*w].g=BinToDec(G[Col+Row*w]);
    IPimage[Col+Row*w].b=BinToDec(B[Col+Row*w]);
    
    

    //Torus
    OPimage[tmp1*w+tmp2]=IPimage[Row*w+Col];

    //printf("1");
    
    //printf("%d %d %d %d\n",Col,Row,tmp1,tmp2);
    //OPimage[tmp1 + n*tmp2]=IPimage[Col+Row*n];
    return;
}



int main(void)
{
    int i, w, h;
    char blah[3];

    FILE *f, *f2;
    
    //clock_t begin = clock();
    //clock_t begin1=clock();

    f=fopen("man.ppm", "rb");
    f2=fopen("testencrypted.ppm", "wb"); 
    fscanf(f, "%s\n", blah);
    fscanf(f, "%d %d\n", &w, &h);
    fscanf(f, "%d\n", &i);
    struct pixel image[h][w]; 
    fread(&image, sizeof(image), 1, f);
    
  
    
    //double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    //printf("%f",time_spent);

    //long R[h][w],G[h][w],B[h][w];
    struct pixel *d_A,*d_F;
    long *d_R,*d_G,*d_B;
    
    long n=w*h;
    //const long size=n;
    const long bytes = 3*sizeof(unsigned char)*n;


    //Assigning memory in device

    hipMalloc((void **)&d_A,sizeof(pixel)*n);
    hipMalloc((void **)&d_F,sizeof(pixel)*n);
    hipMalloc((void **)&d_R,sizeof(long)*n);
    hipMalloc((void **)&d_G,sizeof(long)*n);
    hipMalloc((void **)&d_B,sizeof(long)*n);


    hipMemcpy(d_A,image,bytes,hipMemcpyHostToDevice);

/*  cudaMemcpy(&d_R,&image,bytes,cudaMemcpyHostToDevice);
    cudaMemcpy(&d_G,&image,bytes,cudaMemcpyHostToDevice);
    cudaMemcpy(&d_B,&image,bytes,cudaMemcpyHostToDevice);
  */ 
  //dim3 X((n-1)/4+1,(n-1)/4+1);
   //dim3 Y( 4,4);

   dim3 threadsPerBlock(32, 32);
   dim3 numBlocks(w/threadsPerBlock.x,h/threadsPerBlock.y); 

    clock_t begin = clock();
    PictureKernel<<<numBlocks,threadsPerBlock>>>(d_A,d_F,d_R,d_G,d_B,w,h);    
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("%f micro seconds\n",time_spent*1000000);
    
    hipMemcpy(image,d_F,bytes,hipMemcpyDeviceToHost);



    hipFree(d_A);
    hipFree(d_F);
    hipFree(d_R);
    hipFree(d_G);
    hipFree(d_B);


    


    fprintf(f2, "%s\n", blah);
    fprintf(f2, "%d %d\n", w, h);
    fprintf(f2, "%d\n", 255);
    
    fwrite(&image, sizeof(image), 1, f2);
    
    fclose(f);
    fclose(f2);
    
    return 0;
}

