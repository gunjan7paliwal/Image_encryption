//#include "stdafx.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"



struct pixel
{  
    unsigned char r;
    unsigned char g;
    unsigned char b;
  
};

__device__
int substitution(long *a){
    //key=135
    *a=(*a+135)%256;
    return *a;   
}

__device__
int rev_substitution(long *a){
    *a=(*a-135+256)%256;
    return *a;
}

__device__
int block_cipher(int* a,int index){
    //key= 0->45,1->127,2->5,3->255,5->90
    switch(*a%5){
        case 0: *a=(*a-45)%255;
                break;
        default:break;      
    }
    return *a;
}

__device__
int power(int a,int b){
    long sum=1;
   for (int i=1;i<=b;i++){
       sum=sum*a;     
   }
   return sum; 
}

__device__
void hamming_code(long *a)
{
    long z=*a;
    int d1,d2,d3,d4,d6,d7,d8,p1,p2,p3,tmp1,tmp2;
    tmp2=power(10,7);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d1=tmp2;
    tmp2=power(10,6);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d2=tmp2;
    tmp2=power(10,5);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d3=tmp2;
    tmp2=power(10,4);
    tmp1=z/tmp2;
    tmp2=tmp1%10;
    d4=tmp2;
    tmp1=z/100;
    tmp2=tmp1%10;
    d6=tmp2;
    tmp1=z/10;
    tmp2=tmp1%10;
    d7=tmp2;
    tmp2=(*a)%10;
    d8=tmp2;

    p1=d1^d2^d3;
    p2=d1^d2^d4;
    p3=d1^d3^d4;
    *a=(*a)-100*d6-10*d7-d8+100*p1+10*p2+p3;
return ;
}
__device__
void bit_rotation(long *a)
{
    long z=*a,tmp,tmp1,tmp2,p1,p2,p3;
    tmp1=z/100;
    tmp2=tmp1%10;
    p1=tmp2;
    tmp1=z/10;
    tmp2=tmp1%10;
    p2=tmp2;
    tmp2=*a%10;
    p3=tmp2;
    *a=(*a)-100*p1-10*p2-p3;
    tmp=p1;
    p1=p2;
    p2=p3;
    p3=tmp;
    *a=(*a)+100*p1+10*p2+p3;
    return;
}

__device__
void rev_bit_rotation(long *a)
{
    long z=*a,tmp1,tmp2,p1,p2,p3,tmp;
    tmp1=z/power(10,2);
    tmp2=tmp1%10;
    p1=tmp2;
    tmp1=z/power(10,1);
    tmp2=tmp1%10;
    p2=tmp2;
    tmp2=*a%10;
    p3=tmp2;
    *a=(*a)-100*p1-10*p2-p3;
    tmp=p3;
    p3=p2;
    p2=p1;
    p1=tmp;
    //printf("%d\n",*a);
    *a=(*a)+100*p1+10*p2+p3;
    //printf("%d",*a);
    return;
}


__device__
void Torus_Auromorphism(int *a,int *b,int c)
{
    //k=1
    int x,y;
    x = (*a+*b)%c;
    y= (*a + 2*(*b))%c;
 
    *a=x;*b=y;
    return;
}

__device__
void Anti_Torus(int *a,int *b,int c)
{
    int x,y;
    x=(2*(*a)+(-1)*(*b)+100000*c)%c;
    y=((-1)*(*a)+*b+10000*c)%c;
    //printf("a=%d b=%d x=%d y=%d\n",*a,*b,x,y);
    *a=x;*b=y;
    //if(*a=2&&*b2)
    
    //*a=x;*b=y;
}


__device__
int DecToBinary(int num)
{
    int count=0, remainder, base = 1, binary = 0, no_of_1s = 0;
    
    while (num > 0)
    {
        count++;
        remainder = num % 2;
        if (remainder == 1)
        {
            no_of_1s++;
        }
        binary = binary + remainder * base;
        num = num / 2;
        base = base * 10;

    }
    binary+=100000000;

//    printf("binary = %d\n",binary);
    return binary;
}

__device__
int BinToDec(int num){
    num-=100000000;
    int dec=0,k=1,i=0;
    //printf("%d",num);

    while(1){
        
        dec=dec + k*(num%10);
        if(i==0)
            i=1;
        else
            i=2*i;
        if(num==0)
        {
            break;
        }

        k=2*i;
        
        num/=10;
    }
    //printf("\n%d",dec);
    return dec;
}



__global__
void PictureKernel(struct pixel* IPimage, struct pixel* OPimage,long* R,long* G,long* B,int w,int h)
{
    // Calculate the row # of the d_Pin and d_Pout element
    int Col = blockIdx.x*blockDim.x + threadIdx.x;
    // Calculate the column # of the d_Pin and d_Pout element
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    // each thread computes one element of d_Pout if in range
   // long a,b,c;
    long i=Row*w+Col;
    R[i]=IPimage[i].r;
    G[i]=IPimage[i].g;
    B[i]=IPimage[i].b;


    rev_substitution(&R[i]);rev_substitution(&G[i]);rev_substitution(&B[i]);
    IPimage[i].r=R[i];
    IPimage[i].g=G[i];
    IPimage[i].b=B[i];

    OPimage[Row*w+Col]=IPimage[Col+Row*w];
    
    return;
}



int main(void)
{
    int i, w, h;
    char blah[3];

    FILE *f, *f2;
    
    //clock_t begin = clock();
    //clock_t begin1=clock();

    f=fopen("sub_encrypted.ppm", "rb");
    f2=fopen("sub_decrypted.ppm", "wb"); 
    fscanf(f, "%s\n", blah);
    fscanf(f, "%d %d\n", &w, &h);
    fscanf(f, "%d\n", &i);
    struct pixel image[h][w]; 
    fread(&image, sizeof(image), 1, f);
    
  
    
    //double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    //printf("%f",time_spent);

    //long R[h][w],G[h][w],B[h][w];
    struct pixel *d_A,*d_F;
    long *d_R,*d_G,*d_B;
    
    long n=w*h;
    //const long size=n;
    const long bytes = 3*sizeof(unsigned char)*n;


    //Assigning memory in device

    hipMalloc((void **)&d_A,sizeof(pixel)*n);
    hipMalloc((void **)&d_F,sizeof(pixel)*n);
    hipMalloc((void **)&d_R,sizeof(long)*n);
    hipMalloc((void **)&d_G,sizeof(long)*n);
    hipMalloc((void **)&d_B,sizeof(long)*n);

    hipMemcpy(d_A,image,bytes,hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(w/threadsPerBlock.x,h/threadsPerBlock.y); 

    clock_t begin = clock();
    PictureKernel<<<numBlocks,threadsPerBlock>>>(d_A,d_F,d_R,d_G,d_B,w,h);    
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("%f micro seconds\n",time_spent*1000000);
    
    hipMemcpy(image,d_F,bytes,hipMemcpyDeviceToHost);



    hipFree(d_A);
    hipFree(d_F);
    hipFree(d_R);
    hipFree(d_G);
    hipFree(d_B);


    


    fprintf(f2, "%s\n", blah);
    fprintf(f2, "%d %d\n", w, h);
    fprintf(f2, "%d\n", 255);
    
    fwrite(&image, sizeof(image), 1, f2);
    
    fclose(f);
    fclose(f2);
    
    return 0;
}

